#include "hip/hip_runtime.h"
#include "../shared/globals.hpp"
#include "../shared/timer.hpp"
#include "../shared/argument_parsing.cuh"
#include "../shared/graph.cuh"
#include "../shared/subgraph.cuh"
#include "../shared/partitioner.cuh"
#include "../shared/subgraph_generator.cuh"
#include "../shared/gpu_error_check.cuh"
#include "../shared/gpu_kernels.cuh"
#include "../shared/subway_utilities.hpp"


int main(int argc, char** argv)
{
    hipFree(0);

    ArgumentParser arguments(argc, argv, true, false);
    Timer timer;
    timer.Start();
    Graph<OutEdge> graph(arguments.input, false);
    graph.ReadGraph();
    float readtime = timer.Finish();
    cout << "Graph Reading finished in " << readtime/1000 << " (s).\n";
    for(uint i=0; i<graph.num_nodes; i++)
    {
        graph.value[i] = i;
        graph.label1[i] = true;
        graph.label2[i] = false;
    }


    ull n = graph.num_nodes;
    gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, n * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(graph.d_value, graph.value, n * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(graph.d_label1, graph.label1, n * sizeof(bool), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(graph.d_label2, graph.label2, n * sizeof(bool), hipMemcpyHostToDevice));
    
    Subgraph<OutEdge> subgraph(graph.num_nodes, graph.num_edges);
    SubgraphGenerator<OutEdge> subgen(graph);
    subgen.generate(graph, subgraph);


    Partitioner<OutEdge> partitioner;
    timer.Start();
    uint gItr = 0;
    bool finished;
    bool *d_finished;
    gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
    while (subgraph.numActiveNodes>0)
    {
        gItr++;
        partitioner.partition(subgraph, subgraph.numActiveNodes);
        // a super iteration
        for(int i=0; i<partitioner.numPartitions; i++)
        {
            hipDeviceSynchronize();
            gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdge), hipMemcpyHostToDevice));
            hipDeviceSynchronize();

            //moveUpLabels<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(subgraph.d_activeNodes, graph.d_label, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
            mixLabels<<<partitioner.partitionNodeSize[i]/512 + 1 , 512>>>(subgraph.d_activeNodes, graph.d_label1, graph.d_label2, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
            uint itr = 0;
            do
            {
                itr++;
                finished = true;
                gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
                cc_async<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(partitioner.partitionNodeSize[i],
                                                        partitioner.fromNode[i],
                                                        partitioner.fromEdge[i],
                                                        subgraph.d_activeNodes,
                                                        subgraph.d_activeNodesPointer,
                                                        subgraph.d_activeEdgeList,
                                                        graph.d_outDegree,
                                                        graph.d_value, 
                                                        d_finished,
                                                        (itr%2==1) ? graph.d_label1 : graph.d_label2,
                                                        (itr%2==1) ? graph.d_label2 : graph.d_label1);

                hipDeviceSynchronize();
                gpuErrorcheck( hipPeekAtLastError() );
                gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
            }while(!(finished));
            cout << itr << ((itr>1) ? " Inner Iterations" : " Inner Iteration") << " in Global Iteration " << gItr << ", Partition " << i  << endl;
        }
        subgen.generate(graph, subgraph);
    }
    float runtime = timer.Finish();
    cout << "Processing finished in " << runtime/1000 << " (s).\n";
    gpuErrorcheck(hipMemcpy(graph.value, graph.d_value, n*sizeof(uint), hipMemcpyDeviceToHost));
    utilities::PrintResults(graph.value, min(30, graph.num_nodes));
    if(arguments.hasOutput)
        utilities::SaveResults(arguments.output, graph.value, graph.num_nodes);
}

