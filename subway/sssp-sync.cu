#include "hip/hip_runtime.h"
#include "../shared/globals.hpp"
#include "../shared/timer.hpp"
#include "../shared/argument_parsing.cuh"
#include "../shared/graph.cuh"
#include "../shared/subgraph.cuh"
#include "../shared/partitioner.cuh"
#include "../shared/subgraph_generator.cuh"
#include "../shared/gpu_error_check.cuh"
#include "../shared/gpu_kernels.cuh"
#include "../shared/subway_utilities.hpp"


int main(int argc, char** argv)
{
    hipFree(0);

    ArgumentParser arguments(argc, argv, true, false);
    Timer timer;
    timer.Start();
    Graph<OutEdgeWeighted> graph(arguments.input, true);
    graph.ReadGraph();
    float readtime = timer.Finish();
    cout << "Graph Reading finished in " << readtime/1000 << " (s).\n";
    for(uint i=0; i<graph.num_nodes; i++)
    {
        graph.value[i] = DIST_INFINITY;
        graph.label1[i] = false;
        graph.label2[i] = false;
    }
    graph.value[arguments.sourceNode] = 0;
    graph.label1[arguments.sourceNode] = false;
    graph.label2[arguments.sourceNode] = true;


    ull n = graph.num_nodes;
    gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, n * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(graph.d_value, graph.value, n * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(graph.d_label1, graph.label1, n * sizeof(bool), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(graph.d_label2, graph.label2, n * sizeof(bool), hipMemcpyHostToDevice));
    Subgraph<OutEdgeWeighted> subgraph(graph.num_nodes, graph.num_edges);
    SubgraphGenerator<OutEdgeWeighted> subgen(graph);
    subgen.generate(graph, subgraph);


    Partitioner<OutEdgeWeighted> partitioner;
    timer.Start();
    uint itr = 0;
    while (subgraph.numActiveNodes>0)
    {
        itr++;
        partitioner.partition(subgraph, subgraph.numActiveNodes);
        // a super iteration
        for(int i=0; i<partitioner.numPartitions; i++)
        {
            hipDeviceSynchronize();
            gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdgeWeighted), hipMemcpyHostToDevice));
            hipDeviceSynchronize();

            moveUpLabels<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(subgraph.d_activeNodes, graph.d_label1, graph.d_label2, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);

            sssp_kernel<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(partitioner.partitionNodeSize[i],
                                                    partitioner.fromNode[i],
                                                    partitioner.fromEdge[i],
                                                    subgraph.d_activeNodes,
                                                    subgraph.d_activeNodesPointer,
                                                    subgraph.d_activeEdgeList,
                                                    graph.d_outDegree,
                                                    graph.d_value, 
                                                    //d_finished,
                                                    graph.d_label1,
                                                    graph.d_label2);

            hipDeviceSynchronize();
            gpuErrorcheck( hipPeekAtLastError() );
        }
        subgen.generate(graph, subgraph);
    }
    float runtime = timer.Finish();
    cout << "Processing finished in " << runtime << " (ms).\n";
    cout << "Number of iterations = " << itr << endl;
    gpuErrorcheck(hipMemcpy(graph.value, graph.d_value, n*sizeof(uint), hipMemcpyDeviceToHost));
    utilities::PrintResults(graph.value, min(30, graph.num_nodes));
    if(arguments.hasOutput)
        utilities::SaveResults(arguments.output, graph.value, graph.num_nodes);
}

