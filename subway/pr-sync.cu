#include "hip/hip_runtime.h"
#include "../shared/globals.hpp"
#include "../shared/timer.hpp"
#include "../shared/argument_parsing.cuh"
#include "../shared/graph.cuh"
#include "../shared/subgraph.cuh"
#include "../shared/partitioner.cuh"
#include "../shared/subgraph_generator.cuh"
#include "../shared/gpu_error_check.cuh"
#include "../shared/gpu_kernels.cuh"
#include "../shared/subway_utilities.hpp"
#include "../shared/test.cuh"
#include "../shared/test.cu"


int main(int argc, char** argv)
{
    hipFree(0);

    ArgumentParser arguments(argc, argv, true, false);
    Timer timer;
    timer.Start();
    GraphPR<OutEdge> graph(arguments.input, true);
    graph.ReadGraph();
    float readtime = timer.Finish();
    cout << "Graph Reading finished in " << readtime/1000 << " (s).\n";
    //for(unsigned int i=0; i<100; i++)
    //    cout << graph.edgeList[i].end << " " << graph.edgeList[i].w8;
    float initPR = 0.15;
    float acc = 0.01;
    for(uint i=0; i<graph.num_nodes; i++)
    {
        graph.delta[i] = initPR;
        graph.value[i] = 0;
    }
    //graph.value[arguments.sourceNode] = 0;
    //graph.label[arguments.sourceNode] = true;


    ull n = graph.num_nodes;
    gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, n * sizeof(uint), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(graph.d_value, graph.value, n * sizeof(float), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(graph.d_delta, graph.delta, n * sizeof(float), hipMemcpyHostToDevice));
    Subgraph<OutEdge> subgraph(graph.num_nodes, graph.num_edges);
    SubgraphGenerator<OutEdge> subgen(graph);
    subgen.generate(graph, subgraph, acc);

    Partitioner<OutEdge> partitioner;
    timer.Start();
    uint gItr = 0;
    while (subgraph.numActiveNodes>0)
    {
        gItr++;
        partitioner.partition(subgraph, subgraph.numActiveNodes);
        // a super iteration
        for(int i=0; i<partitioner.numPartitions; i++)
        {
            hipDeviceSynchronize();
            gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdge), hipMemcpyHostToDevice));
            hipDeviceSynchronize();

            pr_kernel<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(partitioner.partitionNodeSize[i],
                                                partitioner.fromNode[i],
                                                partitioner.fromEdge[i],
                                                subgraph.d_activeNodes,
                                                subgraph.d_activeNodesPointer,
                                                subgraph.d_activeEdgeList,
                                                graph.d_outDegree,
                                                graph.d_value,
                                                graph.d_delta,
                                                acc);


            hipDeviceSynchronize();
            gpuErrorcheck( hipPeekAtLastError() );
        }
        subgen.generate(graph, subgraph, acc);
    }
    float runtime = timer.Finish();
    cout << "Processing finished in " << runtime/1000 << " (s).\n";
    cout << "Number of iterations = " << gItr << endl;
    gpuErrorcheck(hipMemcpy(graph.value, graph.d_value, n*sizeof(float), hipMemcpyDeviceToHost));
    utilities::PrintResults(graph.value, min(30, graph.num_nodes));

    if(arguments.hasOutput)
        utilities::SaveResults(arguments.output, graph.value, graph.num_nodes);
}

