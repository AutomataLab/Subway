#include "hip/hip_runtime.h"
#include "subgraph_generator.cuh"
#include "graph.cuh"
#include "subgraph.cuh"
#include "gpu_error_check.cuh"

const unsigned int NUM_THREADS = 64;

const unsigned int THRESHOLD_THREAD = 50000;

__global__ void prePrefix(uint *activeNodesLabeling, uint *activeNodesDegree, 
                          uint *outDegree, bool *label1, bool *label2, uint numNodes)
{
    uint id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < numNodes){
        activeNodesLabeling[id] = label1[id] || label2[id]; // label1 is always zero in sync
        //activeNodesLabeling[id] = label[id];
        //activeNodesLabeling[id] = 1;
        activeNodesDegree[id] = 0;
        if(activeNodesLabeling[id] == 1)
            activeNodesDegree[id] = outDegree[id];
    }
}

__global__ void prePrefix(uint *activeNodesLabeling, uint *activeNodesDegree,
                          uint *outDegree, float *delta, uint numNodes, float acc)
{
    uint id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < numNodes){
        if(delta[id] > acc)
        {
            activeNodesLabeling[id] = 1;
        }
        else
        {
            activeNodesLabeling[id] = 0;
        }
        activeNodesDegree[id] = 0;
        if(activeNodesLabeling[id] == 1)
            activeNodesDegree[id] = outDegree[id];
    }
}

__global__ void makeQueue(uint *activeNodes, uint *activeNodesLabeling,
                          uint *prefixLabeling, uint numNodes)
{
    uint id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < numNodes && activeNodesLabeling[id] == 1){
        activeNodes[prefixLabeling[id]] = id;
    }
}

__global__ void makeActiveNodesPointer(ull *activeNodesPointer, uint *activeNodesLabeling, 
                                       uint *prefixLabeling, ull *prefixSumDegrees, 
                                       uint numNodes)
{
    uint id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < numNodes && activeNodesLabeling[id] == 1){
        activeNodesPointer[prefixLabeling[id]] = prefixSumDegrees[id];
    }
}

// pthread
template <class E>
void dynamic(uint tId,
             uint numThreads,
             uint numActiveNodes,
             uint *activeNodes,
             uint *outDegree, 
             ull *activeNodesPointer,
             ull *nodePointer, 
             E *activeEdgeList,
             E *edgeList)
{

    uint chunkSize = ceil(numActiveNodes / (double)numThreads);
    uint left, right;
    left = tId * chunkSize;
    right = min(left+chunkSize, numActiveNodes);
    uint thisNode;
    uint thisDegree;
    ull fromHere;
    ull fromThere;

    for(uint i=left; i<right; i++)
    {
        thisNode = activeNodes[i];
        thisDegree = outDegree[thisNode];
        fromHere = activeNodesPointer[i];
        fromThere = nodePointer[thisNode];
        for( uint j=0; j<thisDegree; j++)
        {
            activeEdgeList[fromHere+j] = edgeList[fromThere+j];
        }
    }
}

template <class E>
SubgraphGenerator<E>::SubgraphGenerator(Graph<E> &graph)
{
    ull l = graph.num_nodes;
    gpuErrorcheck(hipHostMalloc(&activeNodesLabeling, l * sizeof(uint)));
    gpuErrorcheck(hipHostMalloc(&activeNodesDegree, l * sizeof(uint)));
    gpuErrorcheck(hipHostMalloc(&prefixLabeling, l * sizeof(uint)));
    gpuErrorcheck(hipHostMalloc(&prefixSumDegrees, (l+1) * sizeof(ull)));

    gpuErrorcheck(hipMalloc(&d_activeNodesLabeling, l * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_activeNodesDegree, l * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_prefixLabeling, l * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_prefixSumDegrees , (l+1) * sizeof(ull)));
}

template <class E>
SubgraphGenerator<E>::SubgraphGenerator(GraphPR<E> &graph)
{
    ull l = graph.num_nodes;
    gpuErrorcheck(hipHostMalloc(&activeNodesLabeling, l * sizeof(uint)));
    gpuErrorcheck(hipHostMalloc(&activeNodesDegree, l * sizeof(uint)));
    gpuErrorcheck(hipHostMalloc(&prefixLabeling, l * sizeof(uint)));
    gpuErrorcheck(hipHostMalloc(&prefixSumDegrees, (l+1) * sizeof(ull)));

    gpuErrorcheck(hipMalloc(&d_activeNodesLabeling, l * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_activeNodesDegree, l * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_prefixLabeling, l * sizeof(uint)));
    gpuErrorcheck(hipMalloc(&d_prefixSumDegrees , (l+1) * sizeof(ull)));
}

template <class E>
void SubgraphGenerator<E>::generate(Graph<E> &graph, Subgraph<E> &subgraph)
{
    //std::chrono::time_point<std::chrono::system_clock> startDynG, finishDynG;
    //startDynG = std::chrono::system_clock::now();
    prePrefix<<<graph.num_nodes/512+1, 512>>>(d_activeNodesLabeling, d_activeNodesDegree, graph.d_outDegree, graph.d_label1, graph.d_label2, graph.num_nodes);
    thrust::device_ptr<uint> ptr_labeling(d_activeNodesLabeling);
    thrust::device_ptr<uint> ptr_labeling_prefixsum(d_prefixLabeling);
    subgraph.numActiveNodes = thrust::reduce(ptr_labeling, ptr_labeling + graph.num_nodes);
    //std::cout << "Number of Active Nodes = " << subgraph.numActiveNodes << std::endl;
    thrust::exclusive_scan(ptr_labeling, ptr_labeling + graph.num_nodes, ptr_labeling_prefixsum);
    makeQueue<<<graph.num_nodes/512+1, 512>>>(subgraph.d_activeNodes, d_activeNodesLabeling, d_prefixLabeling, graph.num_nodes);
    gpuErrorcheck(hipMemcpy(subgraph.activeNodes, subgraph.d_activeNodes, subgraph.numActiveNodes*sizeof(uint), hipMemcpyDeviceToHost));
    thrust::device_ptr<uint> ptr_degrees(d_activeNodesDegree);
    thrust::device_ptr<ull> ptr_degrees_prefixsum(d_prefixSumDegrees);
    thrust::exclusive_scan(ptr_degrees, ptr_degrees + graph.num_nodes, ptr_degrees_prefixsum);
    makeActiveNodesPointer<<<graph.num_nodes/512+1, 512>>>(subgraph.d_activeNodesPointer, d_activeNodesLabeling, d_prefixLabeling, d_prefixSumDegrees, graph.num_nodes);
    ull n = subgraph.numActiveNodes;
    gpuErrorcheck(hipMemcpy(subgraph.activeNodesPointer, subgraph.d_activeNodesPointer, n*sizeof(ull), hipMemcpyDeviceToHost));
    ull numActiveEdges = 0;
    if(subgraph.numActiveNodes>0)
        numActiveEdges = subgraph.activeNodesPointer[subgraph.numActiveNodes-1] + graph.outDegree[subgraph.activeNodes[subgraph.numActiveNodes-1]];
    ull last = numActiveEdges;
    gpuErrorcheck(hipMemcpy(subgraph.d_activeNodesPointer+subgraph.numActiveNodes, &last, sizeof(ull), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(subgraph.activeNodesPointer, subgraph.d_activeNodesPointer, (n+1)*sizeof(ull), hipMemcpyDeviceToHost));
    //finishDynG = std::chrono::system_clock::now();
    //std::chrono::duration<double> elapsed_seconds_dyng = finishDynG-startDynG;
    //std::time_t finish_time_dyng = std::chrono::system_clock::to_time_t(finishDynG);
    //std::cout << "Dynamic GPU Time = " << elapsed_seconds_dyng.count() << std::endl;
    //td::chrono::time_point<std::chrono::system_clock> startDynC, finishDynC;
    //startDynC = std::chrono::system_clock::now();
    uint numThreads = NUM_THREADS;

    if(subgraph.numActiveNodes < THRESHOLD_THREAD)
        numThreads = 1;

    thread runThreads[numThreads];
    for(uint t=0; t < numThreads; t++)
    {

        runThreads[t] = thread(dynamic<E>,
                                t,
                                numThreads,
                                subgraph.numActiveNodes,
                                subgraph.activeNodes,
                                graph.outDegree, 
                                subgraph.activeNodesPointer,
                                graph.nodePointer, 
                                subgraph.activeEdgeList,
                                graph.edgeList);

    }
    for(uint t=0; t<numThreads; t++)
        runThreads[t].join();
    //finishDynC = std::chrono::system_clock::now();
    //std::chrono::duration<double> elapsed_seconds_dync = finishDynC-startDynC;
    //std::time_t finish_time_dync = std::chrono::system_clock::to_time_t(finishDynC);
    //std::cout << "Dynamic CPU Time = " << elapsed_seconds_dync.count() << std::endl;
}



template <class E>
void SubgraphGenerator<E>::generate(GraphPR<E> &graph, Subgraph<E> &subgraph, float acc)
{
    //std::chrono::time_point<std::chrono::system_clock> startDynG, finishDynG;
    //startDynG = std::chrono::system_clock::now();
    prePrefix<<<graph.num_nodes/512+1, 512>>>(d_activeNodesLabeling, d_activeNodesDegree, graph.d_outDegree, graph.d_delta, graph.num_nodes, acc);
    thrust::device_ptr<uint> ptr_labeling(d_activeNodesLabeling);
    thrust::device_ptr<uint> ptr_labeling_prefixsum(d_prefixLabeling);
    subgraph.numActiveNodes = thrust::reduce(ptr_labeling, ptr_labeling + graph.num_nodes);
    //cout << "Number of Active Nodes = " << subgraph.numActiveNodes << endl;
    thrust::exclusive_scan(ptr_labeling, ptr_labeling + graph.num_nodes, ptr_labeling_prefixsum);
    makeQueue<<<graph.num_nodes/512+1, 512>>>(subgraph.d_activeNodes, d_activeNodesLabeling, d_prefixLabeling, graph.num_nodes);


    ull n = subgraph.numActiveNodes;

    gpuErrorcheck(hipMemcpy(subgraph.activeNodes, subgraph.d_activeNodes, n*sizeof(uint), hipMemcpyDeviceToHost));
    thrust::device_ptr<uint> ptr_degrees(d_activeNodesDegree);
    thrust::device_ptr<ull> ptr_degrees_prefixsum(d_prefixSumDegrees);
    thrust::exclusive_scan(ptr_degrees, ptr_degrees + graph.num_nodes, ptr_degrees_prefixsum);
    makeActiveNodesPointer<<<graph.num_nodes/512+1, 512>>>(subgraph.d_activeNodesPointer, d_activeNodesLabeling, d_prefixLabeling, d_prefixSumDegrees, graph.num_nodes);
    gpuErrorcheck(hipMemcpy(subgraph.activeNodesPointer, subgraph.d_activeNodesPointer, n*sizeof(ull), hipMemcpyDeviceToHost));
    ull numActiveEdges = 0;
    if(subgraph.numActiveNodes>0)
        numActiveEdges = subgraph.activeNodesPointer[subgraph.numActiveNodes-1] + graph.outDegree[subgraph.activeNodes[subgraph.numActiveNodes-1]];    
    ull last = numActiveEdges;
    gpuErrorcheck(hipMemcpy(subgraph.d_activeNodesPointer+subgraph.numActiveNodes, &last, sizeof(ull), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(subgraph.activeNodesPointer, subgraph.d_activeNodesPointer, (n+1)*sizeof(ull), hipMemcpyDeviceToHost));
    //finishDynG = std::chrono::system_clock::now();
    //std::chrono::duration<double> elapsed_seconds_dyng = finishDynG-startDynG;
    //std::time_t finish_time_dyng = std::chrono::system_clock::to_time_t(finishDynG);
    //std::cout << "Dynamic GPU Time = " << elapsed_seconds_dyng.count() << std::endl;
    //td::chrono::time_point<std::chrono::system_clock> startDynC, finishDynC;
    //startDynC = std::chrono::system_clock::now();
    uint numThreads = NUM_THREADS;

    if(subgraph.numActiveNodes < THRESHOLD_THREAD)
        numThreads = 1;

    thread runThreads[numThreads];
    for(uint t=0; t<numThreads; t++)
    {

        runThreads[t] = thread(dynamic<E>,
                                t,
                                numThreads,
                                subgraph.numActiveNodes,
                                subgraph.activeNodes,
                                graph.outDegree, 
                                subgraph.activeNodesPointer,
                                graph.nodePointer, 
                                subgraph.activeEdgeList,
                                graph.edgeList);

    }
    for(uint t=0; t<numThreads; t++)
        runThreads[t].join();
    //finishDynC = std::chrono::system_clock::now();
    //std::chrono::duration<double> elapsed_seconds_dync = finishDynC-startDynC;
    //std::time_t finish_time_dync = std::chrono::system_clock::to_time_t(finishDynC);
    //std::cout << "Dynamic CPU Time = " << elapsed_seconds_dync.count() << std::endl;
}

template class SubgraphGenerator<OutEdge>;
template class SubgraphGenerator<OutEdgeWeighted>;

